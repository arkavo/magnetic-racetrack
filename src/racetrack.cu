
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include <limits>
#include <string>

enum spin 
{
    UP = 1,
    DOWN = -1    
};

class cell
{
    public:
        spin SPIN;
        double U_H;
    
    void potential(int j,int i)
    {
        U_H = SPIN/abs(j - i);
    }
    
};

class track
{
    public:
        int length;
    
    cell* TRACK_START;

    void set_track()
    {
        TRACK_START = (cell*)malloc(sizeof(cell)*length);
    }
    void assign_cell(int i,cell unit)
    {
        *(TRACK_START+i) = unit;
    }
    void display_track()
    {
        for(int i=0;i<length;i++)
        {
            std::cout<<(*(TRACK_START+i)).SPIN<<"\n";
        }
    }
};

int main(int argc, char* argv[])
{
    int LENGTH = std::stoi(argv[1]);

    cell test;
    test.SPIN = DOWN;
    std::cout<<test.SPIN<<"\n";
    
    track test_track;
    test_track.length = LENGTH;
    test_track.set_track();
    
    for(int i=0;i<LENGTH;i++)
    {
        if(i%2==0)
            test.SPIN = UP;
        else
            test.SPIN = DOWN;
        test_track.assign_cell(i,test);
    }
    test_track.display_track();

    return 0;
}